#include <math_functions.h>
#include "cuDL/util/math_funcs.hpp"


void inner_product_gpu(const CBLAS_TRANSPOSE TransA,
                       const CBLAS_TRANSPOSE TransB,
                       const int M, const int N,
                       const int K, const float alpha,
                       const float *A, const float *B,
                       const float beta, float *C) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;

  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasHandle_t handle;
  hipblasSgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N);
}

